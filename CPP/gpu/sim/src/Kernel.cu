#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 #ifndef _FDTD3DGPUKernel_H_
 #define _FDTD3DGPUKernel_H_
#include "Kernel.cuh"
#include "FDTD3dGPU.h"
#include <hip/hip_runtime_api.h>

// #include <hip/hip_cooperative_groups.h>

// namespace cg = cooperative_groups;


__device__ int getBeta(
int *aux,
int i
){
  return min(aux[i] & (1 << 0), 1);
}

__device__ int getExcitor(
int *aux,
int i
){
  return min(aux[i] & (1 << 1), 1);
}

__device__ int getWall(
int *aux,
int i
){
  return min(aux[i] & (1 << 2), 1);
}

__device__ float pressureStep(
  float *v_x_prev,
  float *v_y_prev,
  float *p_prev,
  int *aux,
  float *sigma,
  int i
)
{
  float divergence = v_x_prev[i] - v_x_prev[i - STRIDE_X] + v_y_prev[i] - v_y_prev[i - STRIDE_Y];
  float p_denom = 1 + (1 - getBeta(aux, i) + sigma[i]) * DT;
  return (p_prev[i] - COEFF_DIVERGENCE * divergence)/p_denom;
}


__global__ void AudioKernel(
  float *v_x_prev,
  float *v_y_prev,
  float *p_prev,
  float *v_x,
  float *v_y,
  float *p,
  int *aux,
  float *sigma,
  float *audioBuffer,
  int iter
)
{
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  int idy=blockIdx.y*blockDim.y+threadIdx.y;

  int i = (idx + PAD_HALF) + STRIDE_Y * (idy + PAD_HALF);

  //PRESSURE------------------------------

  float p_current = pressureStep(v_x_prev, v_y_prev, p_prev, aux, sigma, i);
  float p_right = pressureStep(v_x_prev, v_y_prev, p_prev, aux, sigma, i + STRIDE_X);
  float p_down = pressureStep(v_x_prev, v_y_prev, p_prev, aux, sigma, i + STRIDE_Y);
  p[i] = p_current;

  //VB------------------------------
  //TODO: not sure if this is supposed to be previous or next pressure
  float delta_p = max(P_MOUTH - p_prev[p_bore_index], 0.0f);
  float vb_x = 0;
  float vb_y = 0;
  int wall = getWall(aux, i);
  int excitor = getExcitor(aux, i);
  int wall_down = getWall(aux, i + STRIDE_Y);
  vb_x = excitor * (1 - delta_p / DELTA_P_MAX) * sqrt(2 * delta_p / RHO) * VB_COEFF / num_excite;
  vb_y = wall * ADMITTANCE * p_current + wall_down * -ADMITTANCE * p_down;

  //VELOCITY------------------------------
  int beta_current = getBeta(aux, i);

  float beta_x = min(beta_current, getBeta(aux, i + STRIDE_X));
  float grad_x = p_right - p_current;
  float sigma_prime_dt_x = (1 - beta_x + sigma[i]) * DT;
  v_x[i] = beta_x * v_x_prev[i] - beta_x * beta_x * COEFF_GRADIENT * grad_x + sigma_prime_dt_x * vb_x;

  float beta_y = min(beta_current, getBeta(aux, i + STRIDE_Y));
  float grad_y = p_down - p[i];
  float sigma_prime_dt_y = (1 - beta_y + sigma[i]) * DT;
  v_y[i] = beta_y * v_y_prev[i] - beta_y * beta_y * COEFF_GRADIENT * grad_y + sigma_prime_dt_y * vb_y;

  if(i == listen_index){
    audioBuffer[iter] = p_current;
  }

  
}

#endif