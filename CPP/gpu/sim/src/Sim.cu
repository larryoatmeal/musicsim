#include "Sim.h"


#include <iostream>
#include <algorithm>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "Kernel.cuh"
#include <stdlib.h>

dim3              dimBlock;
dim3              dimGrid;

void SimState::step(){

};

float SimState::read_pressure(){
  
}
SimState::SimState(float *sigma, int * aux_data, int argc, char *argv[]){
  
}
SimState::~SimState(){
  checkCudaErrors(hipFree(bufferP_in));
  checkCudaErrors(hipFree(bufferVx_in));
  checkCudaErrors(hipFree(bufferVy_in));
  checkCudaErrors(hipFree(bufferP_out));
  checkCudaErrors(hipFree(bufferVx_out));
  checkCudaErrors(hipFree(bufferVy_out));
  checkCudaErrors(hipFree(bufferAux_in));
};
int SimState::GetWidth(){

};
int SimState::GetHeight(){

};
float SimState::GetPressure(int x, int y){

};


void SimState::gpu_step(){
  // Launch the kernel
  // printf("launch kernel\n");
  AudioKernel<<<dimGrid, dimBlock>>>(
    bufferVx_in,
    bufferVy_in,
    bufferP_in,
    bufferVx_out,
    bufferVy_out,
    bufferP_out,
    bufferAux_in,
    buffersSigma_in
  );
  // check for error
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

  std::swap<float *>(bufferP_in, bufferP_out);
  std::swap<float *>(bufferVx_in, bufferVx_out);
  std::swap<float *>(bufferVy_in, bufferVy_out);

}

void SimState::read_back(){
    // float * output_from_gpu = (float *) calloc(N_TOTAL, sizeof(float));
    // // Wait for the kernel to complete
    // checkCudaErrors(hipDeviceSynchronize());
    // // Read the result back, result is in bufferP_in (after final toggle)
    // checkCudaErrors(hipMemcpy(output_from_gpu, bufferP_in, size, hipMemcpyDeviceToHost));
}

void SimState::init(float *sigma, int * aux_data, int argc, char *argv[]){
  float * empty = (float *) calloc(N_TOTAL, sizeof(float));

  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, 0);

  checkCudaErrors(hipSetDevice(targetDevice));
  // Allocate memory buffers

  int size = N_TOTAL * sizeof(float);

  checkCudaErrors(hipMalloc((void **)&bufferP_in, size));
  checkCudaErrors(hipMalloc((void **)&bufferVx_in, size));
  checkCudaErrors(hipMalloc((void **)&bufferVy_in, size));
  checkCudaErrors(hipMalloc((void **)&bufferP_out,  size));
  checkCudaErrors(hipMalloc((void **)&bufferVx_out, size));
  checkCudaErrors(hipMalloc((void **)&bufferVy_out, size));
  checkCudaErrors(hipMalloc((void **)&buffersSigma_in, size));
  checkCudaErrors(hipMalloc((void **)&bufferAux_in, N_TOTAL * sizeof(int)));

  checkCudaErrors(hipMemcpy(bufferP_in, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferVx_in, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferVy_in, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferP_out, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferVx_out, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferVy_out, empty, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(buffersSigma_in, sigma, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(bufferAux_in, aux_data, N_TOTAL * sizeof(int), hipMemcpyHostToDevice));

  
  dimBlock.x = 16;
  dimBlock.y = 16;
  dimGrid.x  = W/dimBlock.x;
  dimGrid.y  = H/dimBlock.y; //for now assume this is perfect division
  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);      
};


